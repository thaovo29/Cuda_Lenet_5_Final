#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define M_CONST 16
#define C_CONST 4
#define K_CONST 7
#define TILE_WIDTH 16

#define CHECK(call)\
{\
    const hipError_t error = call;\
    if (error != hipSuccess)\
    {\
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
        fprintf(stderr, "code: %d, reason: %s\n", error,\
                hipGetErrorString(error));\
        exit(EXIT_FAILURE);\
    }\
}

__constant__ float kernelData[M_CONST * C_CONST * K_CONST * K_CONST];

__global__ void conv_forward_kernel(float *y, const float *x, const float *k, const int B, const int M, const int C, const int H, const int W, const int K)
{
    const int H_out = H - K + 1;
    const int W_out = W - K + 1;

#define y4d(i3, i2, i1, i0) y[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
#define x4d(i3, i2, i1, i0) x[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
#define k4d(i3, i2, i1, i0) kernelData[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    int H_grid = ceil(float(H_out) / TILE_WIDTH);
    int W_grid = ceil(float(W_out) / TILE_WIDTH); 
    
    int b = blockIdx.x;                 // batch number
    int m = blockIdx.y;                 // output feature
    int h = (blockIdx.z / W_grid) * TILE_WIDTH + threadIdx.y; // row of the image matrix
    int w = (blockIdx.z % W_grid) * TILE_WIDTH + threadIdx.x; // col of the image matrix

    if (h < H_out && w < W_out) 
    {
        float sum = 0.0f;
        for(int c=0; c<C; c++)             // sum over all input features
        {
            for(int p=0; p<K; p++)         // KxK filter 
                for(int q=0; q<K; q++)
                    sum += x4d(b, c, h+p, w+q) * k4d(m, c, p, q); // 4 dimensions macro resolve thread index
        }
        y4d(b,m,h,w) = sum;
    }

    #undef y4d
    #undef x4d
    #undef k4d
}

__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_y, const float *host_x, const float *host_k, float **device_y_ptr, float **device_x_ptr, float **device_k_ptr, const int B, const int M, const int C, const int H, const int W, const int K)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    const int H_out = H - K + 1;
    const int W_out = W - K + 1;

    int inputSize  = B * C * H * W * sizeof(float);  // input features map is C
    int outputSize = B * M * H_out * W_out * sizeof(float); // output feature map is M
    int maskSize = M * C * K * K * sizeof(float); // C * M filter Maps of size K*K

    CHECK(hipMalloc((void **) device_x_ptr, inputSize));
    CHECK(hipMalloc((void **) device_y_ptr, outputSize));

    // Copy Inout data to device
    CHECK(hipMemcpy(*device_x_ptr, host_x, inputSize, hipMemcpyHostToDevice));
    // Copy Mask data to device
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(kernelData), host_k, maskSize));

}


__host__ void GPUInterface::conv_forward_gpu(float *device_y, const float *device_x, const float *device_k, const int B, const int M, const int C, const int H, const int W, const int K)
{
    // Set the kernel dimensions and call the kernel
    std::cout << "Constant-stream" << std::endl;

    const int H_out = H - K + 1;
    const int W_out = W - K + 1;

    // Create stream
    const int nStreams = 5;

    hipStream_t stream[nStreams];
    for (int i = 0; i < nStreams; i ++)
    {
        CHECK(hipStreamCreate(&stream[i]));
    }
    const int tile_batch_size = 50;

    int H_grid = ceil(float(H_out) / TILE_WIDTH);
    int W_grid = ceil(float(W_out) / TILE_WIDTH);
    int Z = H_grid * W_grid;

    // Block size
    dim3 blockSize(TILE_WIDTH, TILE_WIDTH, 1);

    // Grid size
    dim3 gridSize(M, Z, tile_batch_size);

    static float* d_in;
    static float* d_out;

    hipMalloc((void**)&d_in,sizeof(float) * B * C * H * W);
    hipMalloc((void**)&d_out,sizeof(float) * B * M * H_out * W_out);

    for(int b = 0; b < ceil(float(B / (nStreams * tile_batch_size)));b++)
    {
        for(int i = 0; i < nStreams; i++)
        {
            int offset_in = (i + b * nStreams) * tile_batch_size * C * H * W;
            hipMemcpyAsync(d_in + offset_in, device_x + offset_in, sizeof(float) * tile_batch_size * C * H * W, hipMemcpyHostToDevice, stream[i]);
        }

        CHECK(hipDeviceSynchronize());

        for(int i = 0; i < nStreams; i++)
        {
            int offset_in = (i + b * nStreams) * tile_batch_size * C * H * W;
            int offset_out = (i + b * nStreams) * tile_batch_size* M * H_out * W_out;
            conv_forward_kernel<<<gridSize, blockSize, 0, stream[i]>>>(d_out + offset_out, d_in + offset_in, device_k, tile_batch_size, M, C, H, W, K);
        }

        CHECK(hipDeviceSynchronize());

        for(int i = 0; i < nStreams; i++)
        {
            int offset_out = (i + b*nStreams)*tile_batch_size * M * H_out * W_out;
            hipMemcpyAsync((float*)device_y + offset_out, d_out + offset_out, sizeof(float)*tile_batch_size* M * H_out * W_out, hipMemcpyDeviceToHost, stream[i]);
        }
    }

    CHECK(hipDeviceSynchronize());
    for (int i = 0; i < nStreams; i++) 
        hipStreamDestroy(stream[i]);
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_y, float *device_y, float *device_x, float *device_k, const int B, const int M, const int C, const int H, const int W, const int K)
{
    // Copy the output back to host
    
    const int H_out = H - K + 1;
    const int W_out = W - K + 1;

    int outputSize = B * M * H_out * W_out * sizeof(float);

    CHECK(hipMemcpy(host_y, device_y, outputSize, hipMemcpyDeviceToHost));

    // Free device memory
    CHECK(hipFree(device_x));
    CHECK(hipFree(device_y));
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
